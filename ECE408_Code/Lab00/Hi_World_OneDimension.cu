
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hiWorld() {
    printf("Hi World from the GPU!\n(block.x %d, thread.x %d)\n", blockIdx.x, threadIdx.x);
}

int main(){
    hiWorld<<<4,2>>>();
    //4个网格，每个网格2个线程
    //4*2=8个线程，所以会输出8次
    hipDeviceSynchronize();
    return 0;
}

/*
结果：

Hi World from the GPU!
(block.x 1, thread.x 0)
Hi World from the GPU!
(block.x 1, thread.x 1)
Hi World from the GPU!
(block.x 0, thread.x 0)
Hi World from the GPU!
(block.x 0, thread.x 1)
Hi World from the GPU!
(block.x 2, thread.x 0)
Hi World from the GPU!
(block.x 2, thread.x 1)
Hi World from the GPU!
(block.x 3, thread.x 0)
Hi World from the GPU!
(block.x 3, thread.x 1)

*/
