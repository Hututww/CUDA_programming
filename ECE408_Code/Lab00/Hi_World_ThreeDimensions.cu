
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hiWorld() {
    const int blockIndexForX = blockIdx.x;
    const int threadIndexForX = threadIdx.x;
    const int threadIndexForY = threadIdx.y;
    const int threadIndexForZ = threadIdx.z;
    printf("Hi World from the GPU!\nfrom block.x %d, thread.( %d, %d, %d)\n", blockIndexForX, threadIndexForY, threadIndexForY, threadIndexForZ);
}

int main(){
    const dim3 block_size(3, 4, 5);
    hiWorld<<<1, block_size>>>();
    hipDeviceSynchronize(); 
    return 0;
}

/*

结果：共有 3 * 4 * 5 = 60 个线程

Hi World from the GPU!
from block.x 0, thread.( 2, 2, 2)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 2)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 2)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 2)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 3)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 3)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 3)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 3)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 3)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 3)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 3)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 3)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 3)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 3)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 3)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 3)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 4)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 4)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 4)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 4)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 4)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 4)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 4)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 4)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 4)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 4)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 4)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 4)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 0)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 0)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 0)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 0)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 0)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 0)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 0)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 0)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 0)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 0)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 0)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 0)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 1)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 1)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 1)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 1)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 1)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 1)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 1)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 1)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 1)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 1)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 1)
Hi World from the GPU!
from block.x 0, thread.( 3, 3, 1)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 2)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 2)
Hi World from the GPU!
from block.x 0, thread.( 0, 0, 2)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 2)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 2)
Hi World from the GPU!
from block.x 0, thread.( 1, 1, 2)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 2)
Hi World from the GPU!
from block.x 0, thread.( 2, 2, 2)

*/